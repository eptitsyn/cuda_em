#include "hip/hip_runtime.h"
/*
* Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include <helper_string.h>

#define MAX_DEPTH       16
#define INSERTION_SORT  32

////////////////////////////////////////////////////////////////////////////////
// Selection sort used when depth gets too big or the number of elements drops
// below a threshold.
////////////////////////////////////////////////////////////////////////////////
__device__ void selection_sort(double *data, int left, int right)
{
    for (int i = left ; i <= right ; ++i)
    {
        double min_val = data[i];
        int min_idx = i;

        // Find the smallest value in the range [left, right].
        for (int j = i+1 ; j <= right ; ++j)
        {
            double val_j = data[j];

            if (val_j < min_val)
            {
                min_idx = j;
                min_val = val_j;
            }
        }

        // Swap the values.
        if (i != min_idx)
        {
            data[min_idx] = data[i];
            data[i] = min_val;
        }
    }
}

////////////////////////////////////////////////////////////////////////////////
// Very basic quicksort algorithm, recursively launching the next level.
////////////////////////////////////////////////////////////////////////////////
__global__ void cdp_simple_quicksort(double *data, int left, int right, int depth)
{
    // If we're too deep or there are few elements left, we use an insertion sort...
    if (depth >= MAX_DEPTH || right-left <= INSERTION_SORT)
    {
        selection_sort(data, left, right);
        return;
    }

    double *lptr = data+left;
    double *rptr = data+right;
    double pivot = data[(left+right)/2];

    // Do the partitioning.
    while (lptr <= rptr)
    {
        // Find the next left- and right-hand values to swap
        double lval = *lptr;
        double rval = *rptr;

        // Move the left pointer as long as the pointed element is smaller than the pivot.
        while (lval < pivot)
        {
            lptr++;
            lval = *lptr;
        }

        // Move the right pointer as long as the pointed element is larger than the pivot.
        while (rval > pivot)
        {
            rptr--;
            rval = *rptr;
        }

        // If the swap points are valid, do the swap!
        if (lptr <= rptr)
        {
            *lptr++ = rval;
            *rptr-- = lval;
        }
    }

    // Now the recursive part
    int nright = rptr - data;
    int nleft  = lptr - data;

    // Launch a new block to sort the left part.
    if (left < (rptr-data))
    {
        hipStream_t s;
        hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s >>>(data, left, nright, depth+1);
        hipStreamDestroy(s);
    }

    // Launch a new block to sort the right part.
    if ((lptr-data) < right)
    {
        hipStream_t s1;
        hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
        cdp_simple_quicksort<<< 1, 1, 0, s1 >>>(data, nleft, right, depth+1);
        hipStreamDestroy(s1);
    }
}

