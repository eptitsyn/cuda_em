#include "hip/hip_runtime.h"
//master  2
#include "hip/hip_runtime.h"
#include ""

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <stdio.h>
#include <cstdlib>
#include <time.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <vector>
#include <numeric>
#include "qsort.cu"

using namespace std;

#define M_SQ2PI 2.506628274631000502416
#define M_SQPId2 1.253314137315500251208
#define k 10
#define MAX_ITERATIONS 10
#define TOLERANCE 0.01
#define WINDOW_LENGTH 1040

static void HandleError(hipError_t err,
                        const char* file,
                        int line)
{
	if (err != hipSuccess)
	{
		printf("%s in %s at line %d\n", hipGetErrorString(err),
		       file, line);
		exit(EXIT_FAILURE);
	}
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

typedef struct
{
	int width;
	int height;
	double* elements;
} Matrix;

// Get a matrix element
__device__ double GetElement(const Matrix A, int row, int col)
{
	return A.elements[row * A.width + col];
}

// Set a matrix element
__device__ void SetElement(Matrix A, int row, int col, double value)
{
	A.elements[row * A.width + col] = value;
}

__device__ double* GetSubData(double* data, int i)
{
	return &data[i];
}

__global__ void initCurand(hiprandState *state, unsigned long seed) {
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	hiprand_init(seed, idx, 0, &state[idx]);
}


__device__ int multinom(double r, double* p, int n)
{
	//double r = devrandomdouble();
	double s = 0;
	int i;
	for (i = 0; i < n; ++i)
	{
		s += p[i];
		if (s - r >= 0) break;
	}
	return i;
}

double* readfromfile(const char* DATA_FILENAME, int data_length)
{
	ifstream inFile;
	inFile.open(DATA_FILENAME);
	if (!inFile) {
		cerr << "Unable to open file datafile";
		exit(1);   // call system to stop
	}
	double *data = (double*)malloc(data_length * sizeof(double));
	double x;
	for (int i = 0; i < data_length; ++i)
	{
		inFile >> x;
		data[i] = x;
	}
	inFile.close();
	return data;
};

double std_dev(double* data, int data_length)
{
	double sum = 0;
	for (int i = 0; i < data_length; ++i)
	{
		sum += data[i];
	}
	double mean = sum / (double)data_length;
	double differ;
	double varsum = 0;
	for (int i = 0; i < data_length; ++i)
	{
		differ = data[i] - mean;
		varsum += pow(differ, 2);
	}
	double Variance = varsum / (double)data_length;
	return sqrt(Variance);
}

double randomdouble()
{
	double r = (double)rand() / (double)RAND_MAX;
	return r;
}


void set_initial_guess(double* data, int data_length, double* theta)
{
	//pi
	for (int i = 0; i < k; ++i)
	{
		theta[i] = randomdouble() * 0.9 + 0.1;
	}
	double tsum = 0;
	for (int i = 0; i < k; ++i)
	{
		tsum += theta[i];
	}
	for (int i = 0; i < k; ++i)
	{
		theta[i] /= tsum;
	}
	/*
	for (int i = 0; i < k; ++i)
	{
		theta[i] = theta[i] * 0.9 + 0.1;
	}*/
	//mu
	for (int i = 0; i < k; ++i)
	{
		theta[i + k] = 0;
	}
	//sigma
	for (int i = 0; i < k; ++i)
	{
		theta[i + k * 2] = randomdouble() * 1.5 + 0.25 * std_dev(data, data_length);
	}
}

__device__ double normpdf(double data, double mu, double sigma){
	return exp(-(pow(data - mu, 2)) / (2 * pow(sigma, 2))) / (M_SQ2PI * sigma);
}

__global__ void e_step1(double* glob_data, int data_off, double* theta, int theta_off, Matrix w)
{
	double* pi = &theta[theta_off];
	double* mu = &theta[theta_off + k];
	double* sigma = &theta[theta_off + 2 * k];
	double* data = &glob_data[data_off];

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < w.width && j < w.height)
		if (sigma[j] != 0)
		{
			SetElement(w, j, i, pi[j] * normpdf(data[i], mu[j], sigma[j]) );
		}
		else
		{
			SetElement(w, j, i, data[i] == mu[j] ? pi[j] : 0);
		}
}

__global__ void e_step2(Matrix w)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < w.width)
	{
		double wsum = 0;
		for (int e = 0; e < w.height; ++e)
		{
			wsum += GetElement(w, e, i);
		}
		for (int e = 0; e < w.height; ++e)
		{
			if (wsum != 0)
				SetElement(w, e, i, GetElement(w, e, i) / wsum);
			else
				SetElement(w, e, i, 0);
		}
	}
}

//__device__ int compare(const void * a, const void * b)
//{
//	double fa = *(const double*)a;
//	double fb = *(const double*)b;
//	return (fa > fb) - (fa < fb);
//}



//__device__ int qpart(double* A, int lo, int hi)
//{
//	double pivot = A[lo + (hi - lo) / 2];
//	int i = lo - 1;
//	int j = hi + 1;
//
//	while(1)
//	{
//		i++;
//		while (A[i]<pivot)
//		{
//			j--;
//		}
//		while (A[j]>pivot)
//		{
//			if (i >= j) return j;
//		}
//		double tmp = A[i];
//		A[i] = A[j];
//		A[j] = A[i];
//	}
//}
//__device__ void q1sort(double* A, int lo, int hi)
//{
//	if (lo < hi)
//	{
//		int p = qpart(A, lo, hi);
//		q1sort(A, lo, p);
//		q1sort(A, p + 1, hi);
//	}
//}
//
//__device__ void quiksort(double* base, size_t num)
//{
//	q1sort(base, 0, num - 1);
//}


__global__ void m_step(double* glob_data, int data_off, double* theta, int theta_off, Matrix w, int* y, int* v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;


	if (j < w.height)
	{
		double* pi = &theta[theta_off];
		double* mu = &theta[theta_off + k];
		double* sigma = &theta[theta_off + 2 * k];
		double* data = &glob_data[data_off];
		if(v[j] != 0){

			double* ss = new double[w.width];
			size_t ss_cnt = 0;
			for (int i = 0; i < w.width; ++i)
			{
				if (y[i] == j) {
					ss[ss_cnt] = GetElement(w, j, i);
					ss_cnt++;
				}
			}

			//sort ss
			cdp_simple_quicksort<<<1,1>>>(ss, 0, ss_cnt-1, 0);
			//pis
			pi[j] = v[j] / (double)w.width;

			//mu
			if (v[j] % 2 == 0)
			{
				mu[j] = 0.5*(ss[v[j] / 2 - 1] + ss[v[j] / 2]);
			}
			else
			{
				mu[j] = ss[v[j] / 2];
			}
			//sigma
			double bs = 0;
			for (int i = 0; i < v[j]; ++i)
			{
				bs += abs(ss[i]-mu[j]);
			}
			bs /= v[j];
			sigma[j] = M_SQPId2 * bs;
			delete(ss);
		} else {
			pi[j] = 0;
			mu[j] = 0;
			sigma[j] = 0;
		}
	}

	/*old
	if (j < w.height)
	{
		pi[j] = 0;
		mu[j] = 0;
		sigma[j] = 0;
		for (int e = 0; e < w.width; e++)
		{
			pi[j] += GetElement(w, j, e);
		}
		if (pi[j] != 0) {
			for (int e = 0; e < w.width; e++)
			{
				mu[j] += GetElement(w, j, e) * data[e];
			}
			mu[j] /= pi[j];


			for (int e = 0; e < w.width; e++)
			{
				sigma[j] += GetElement(w, j, e) * powf(data[e] - mu[j], 2);
			}
			sigma[j] /= pi[j];
			sigma[j] = sqrtf(sigma[j]);

			pi[j] /= w.width;
		}
	}
	*/
}

__global__ void s_step(Matrix w, int* y, double* random)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i < w.width)
	{
		double vv[k];
		double sum = 0;
		for (int j = 0; j < k; ++j)
		{
			vv[j]=GetElement(w, j, i);
			sum += vv[j];
		}
		for (int j = 0; j < k; ++j)
		{
			vv[j] /= sum;
		}
		y[i] = multinom(random[i], vv, k);
	}
}

__global__ void s_step2(Matrix w, int* y, int* v)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (j < w.height)
	{
		v[j] = 0;
		for (int i = 0; i < w.width; ++i)
		{
			if (y[i] == j)
			{
				v[j]++;
			}
		}
	}
}

__global__ void compute_ll(double* glob_data, int data_off, double* theta, int theta_off, Matrix w, int* y, int* v, double* ll, double* ll2)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	if (i < w.width)
	{
		double* pi = &theta[theta_off];
		double* mu = &theta[theta_off + k];
		double* sigma = &theta[theta_off + 2 * k];
		double* data = &glob_data[data_off];

		int jj = y[j];
		double llsum = pi[jj] * normpdf(data[i], mu[jj], sigma[jj]);
		ll[i] = log( llsum );
		ll2[i] = llsum;
	}
}

__global__ void compute_ll2(Matrix w, double* ll)
{
	double llsum = 0;
	for (int e = 0; e < w.width; e++)
	{
		llsum += ll[e];
	}
	ll[0] = llsum;
}


hipError_t em_algorithm(double* d_data, int data_off, const int data_length, double* d_theta, int theta_offset, double* h_theta,  bool debug)
{
	double* d_theta_loc = &d_theta[theta_offset];
	double* h_theta_loc = &h_theta[theta_offset];
	//size_t theta_size = ((data_length - window_size) / window_step )* 3 * k * sizeof(double);
	size_t theta_loc_size = 3 * k * sizeof(double);
	Matrix d_W;
	d_W.width = data_length;
	d_W.height = k;
	HANDLE_ERROR(hipMalloc(&d_W.elements, d_W.width * d_W.height * sizeof(double)));

	double* d_ll;
	double* d_ll2;
	double* h_ll = (double*)malloc(sizeof(double) * data_length);
	double* h_ll2 = (double*)malloc(sizeof(double) * data_length);
	HANDLE_ERROR(hipMalloc(&d_ll, data_length * sizeof(double)));
	HANDLE_ERROR(hipMalloc(&d_ll2, data_length * sizeof(double)));

	hiprandGenerator_t rand_gen;
	hiprandCreateGenerator(&rand_gen, HIPRAND_RNG_PSEUDO_DEFAULT);
	hiprandSetPseudoRandomGeneratorSeed(rand_gen, time(NULL));

	int *d_y, *d_v;
	double *d_random;
	HANDLE_ERROR(hipMalloc(&d_random, data_length * sizeof(double)));
	HANDLE_ERROR(hipMalloc(&d_y, data_length * sizeof(int)));
	HANDLE_ERROR(hipMalloc(&d_v, data_length * sizeof(int)));

	dim3 dimBlock(16, k);
	dim3 dimGrid(data_length / dimBlock.x + 1, 1);

	dim3 dimBlocke2(16, 1);
	dim3 dimGride2(data_length / dimBlocke2.x + 1, 1);

	dim3 dimBlockM(1, k);

	dim3 dimBlockLL(16, 1);
	dim3 dimGridLL(data_length / dimBlockLL.x + 1, 1);


	double ll_old = 0;
	for (int i = 0; i < MAX_ITERATIONS; i++)
	{
		printf("iter = %d, ", i);
		e_step1 << <dimGrid, dimBlock >> >(d_data, data_off, d_theta, theta_offset, d_W);
		e_step2 << <dimGride2, dimBlocke2 >> >(d_W);
		//random
		hiprandGenerateUniformDouble(rand_gen, d_random, data_length);
		//
		s_step << <dimGridLL, dimBlockLL >> >(d_W, d_y, d_random);
		s_step2<<<1, dimBlockM>>>(d_W, d_y, d_v);

		int* h_v = (int*)malloc(k * sizeof(int));
		// hipMemcpy(h_v, d_v, k*sizeof(int), hipMemcpyDeviceToHost);
		// for (int i = 0; i < k; ++i)
		// {
		// 	cout << h_v[i] << ", ";
		// }
		// cout << endl;

		m_step << <1, dimBlockM >> >(d_data, data_off, d_theta, theta_offset, d_W, d_y, d_v);
		hipDeviceSynchronize();

//(double* glob_data, int data_off, double* theta, int theta_off, Matrix w, int* y, int* v double* ll, double* ll2)

		compute_ll << <dimGridLL, dimBlockLL >> >(d_data, data_off, d_theta, theta_offset, d_W, d_y, d_v, d_ll, d_ll2);
		compute_ll2 << <1, 1 >> >(d_W, d_ll);
		hipDeviceSynchronize();

		HANDLE_ERROR(hipMemcpy(h_theta_loc, d_theta_loc, theta_loc_size, hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(h_ll, d_ll, data_length * sizeof(double), hipMemcpyDeviceToHost));
		HANDLE_ERROR(hipMemcpy(h_ll2, d_ll2, data_length * sizeof(double), hipMemcpyDeviceToHost));

		double ll_new = h_ll[0];

		printf("ll = %f;\n", ll_new);
		/*if (!isnormal(ll_new))
		{
			hipMemcpy(h_theta_loc, d_theta_loc, theta_loc_size, hipMemcpyDeviceToHost);
			printf("\n");
			for (int i =0 ; i < 3; ++i)
			{
				for (int j = 0; j < k; ++j)
				{
					printf("%f, ", h_theta_loc[i*k+j]);
				}
				printf("\n");
			}
			printf("\n");
		}*/
		//printf("ll = %f;\n", ll_new);
		// if (isnan(ll_new))
		// {
		// 	HANDLE_ERROR(hipMemcpy(h_theta_loc, d_theta_loc, 3*k* sizeof(double), hipMemcpyDeviceToHost));
		// 	for (int i = 0; i < k * 3; ++i)
		// 	{
		// 		printf("%f, ", h_theta_loc[i]);
		// 		(i + 1) % 10 == 0 ? printf("\n") : printf("");
		// 	}
		// 	printf("\n");
		// 	/*
		// 	for (int i = 0; i < data_length; ++i)
		// 	{
		// 		printf("%f, ", h_ll2[i]);
		// 	}
		// 	printf("\n\nW= \n");

		// 	*/
		// 	Matrix h_W;
		// 	h_W.width = data_length;
		// 	h_W.height = k;
		// 	h_W.elements = (double*)malloc(h_W.width * h_W.height * sizeof(double));
		// 	hipMemcpy(h_W.elements, d_W.elements, d_W.width * d_W.height * sizeof(double), hipMemcpyDeviceToHost);

		// 	for (int i = 0; i < data_length * k * 3; i++)
		// 	{
		// 		printf("%f, ", h_W.elements[i]);
		// 		(i + 1) % 10 == 0 ? printf("\n") : printf("");
		// 	}
		// 	printf("\n");

		// 	exit(1);
		// }
		if (abs(ll_new - ll_old) < TOLERANCE)
		{
			printf("end em step %d", i);
			break;
		}
		ll_old = ll_new;
		hipDeviceSynchronize();

		if (debug)
		{
			hipMemcpy(h_theta_loc, d_theta_loc, theta_loc_size, hipMemcpyDeviceToHost);
			for (int i = 0; i < 3; ++i)
			{
				for (int j = 0; j < k; ++j)
				{
					printf("%f, ", h_theta_loc[i * k + j]);
				}
				printf("\n");
			}
			printf("\n");
		}
	}

	hipFree(d_ll);
	hipFree(d_ll2);
	hipFree(d_W.elements);
	hipFree(d_random);
	hipFree(d_y);
	hipFree(d_v);
	return hipSuccess;
}

__global__ void copythetatonext(double *theta, int theta_offset, int theta_length)
{
		for (int j = theta_offset; j < theta_offset+theta_length; ++j)
		{
			theta[j+theta_length] = theta[j];
		}
}

hipError_t slsalgorithm(double* h_data, const int data_length, double* h_theta, const int window_size, const int window_step, const int generate_theta_each_step)
{
	double* d_data = 0;
	double* d_theta = 0;
	int theta_offset = 0;
	int data_off = 0;

	const int steps = (data_length - window_size + 1) / window_step;
	size_t theta_size = (steps * k * 3 * sizeof(double));


	hipError_t cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	HANDLE_ERROR(hipMalloc(&d_theta, theta_size));
	HANDLE_ERROR(hipMalloc(&d_data, data_length * sizeof(double)));

	///
	//set inint guess
	switch (generate_theta_each_step)
	{
	default:
		{
			set_initial_guess(h_data, window_size, h_theta);
			for (int i = 1; i < (steps); ++i)
			{
				for (int j = 0; j < k * 3; ++j)
				{
					h_theta[i * k * 3 + j] = h_theta[j];
				}
			}
			break;
		}
	case 1:
		{
			for (int i = 0; i < steps; ++i)
			{
				set_initial_guess(&h_data[i * window_step], window_size, &h_theta[i * 3 * k]);
			}
			break;
		}
	case 2:
		{
			set_initial_guess(h_data, window_size, h_theta);
			break;
		}
	}

	hipDeviceSynchronize();
	HANDLE_ERROR(hipMemcpy(d_theta, h_theta, theta_size, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(d_data, h_data, data_length * sizeof(double), hipMemcpyHostToDevice));
	/*for (int i = 0; i < k * 3 * steps; ++i)
	{
		h_theta[i] = 0;
	}*/
	/*
	 *
	 */

	for (int i = 0; i < steps; ++i)
	{
		time_t rawtime;
		time(&rawtime);
		struct tm* timeinfo = localtime(&rawtime);

		char* foo = asctime(timeinfo);
		foo[strlen(foo) - 1] = 0;
		printf("[%s] Start EM %d: \n", foo , i);
		data_off = i * window_step;
		theta_offset = i * k * 3;
		em_algorithm(d_data, data_off, window_size, d_theta, theta_offset, h_theta, false);
		if (generate_theta_each_step == 2 && i!=steps-1)
		{
			copythetatonext<<<1,1>>>(d_theta, theta_offset, k * 3);
		}

	}
	hipDeviceSynchronize();
	HANDLE_ERROR(hipMemcpy(h_theta, d_theta, theta_size, hipMemcpyDeviceToHost));
	/*
	 *
	 */
Error:

	hipFree(d_data);
	hipFree(d_theta);
	return cudaStatus;
}

void savetofile(double* theta, int size)
{
	ofstream ofile("output.data");
	if (ofile.is_open())
	{
		for (int i = 0; i < size; ++i)
		{
			for (int j = 0; j < k * 3; ++j)
			{
				ofile << theta[i * k * 3 + j] << ", ";
			}
			if (i != size - 1)
				ofile << endl;
		}
		ofile.close();
	}
	else
		cout << "Unable to open file";
}

int main()
{
	HANDLE_ERROR(hipDeviceReset());
	srand(time(NULL));
	const int data_length = 1100;
	const char* data_filename = "..//data//data_imoex_180323_180424_5min.txt";
	const int window_length = WINDOW_LENGTH;
	const int window_step = 1;
	const int generate_theta_each_step = 0;
	/* 0 - ������������ ���� ��� ����������� � ���������� ��� �� ��� ��������
	 * 1 - ������������ ��� ����� ��� ���� ��������
	 * 2 - ������������ ���������� ��������� ��� ��������� �����������
	 */
	const int steps = (data_length - window_length + 1) / window_step;

	double* data = readfromfile(data_filename, data_length);
	double* theta = (double*)malloc(steps * 3 * k * sizeof(double));

	hipError_t cudaStatus = slsalgorithm(data, data_length, theta, window_length, window_step, generate_theta_each_step);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	savetofile(theta, steps);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}